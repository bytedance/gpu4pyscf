/* Copyright 2023 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS        32
#define SQRT2_PI       0.7978845608028654
#define SQRT_PI        1.7724538509055159

__global__
static void _pcm_d_s(double *matrix_d, double *matrix_s,
                    const double *coords, const double *norm_vec, const double *r_vdw,
                    const double *charge_exp, const double *switch_fun,
                    int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n || j >= n){
        return;
    }

    // calculate xi
    double ei = charge_exp[i];
    double ej = charge_exp[j];
    double xi_ij = ei * ej / sqrt(ei*ei + ej*ej);

    // calculate r
    double dx = coords[3*i]   - coords[3*j];
    double dy = coords[3*i+1] - coords[3*j+1];
    double dz = coords[3*i+2] - coords[3*j+2];
    double rij = norm3d(dx, dy, dz);

    double xi_r_ij = xi_ij * rij;
    if (i == j) rij = 1.0;
    double s = erf(xi_r_ij) / rij;
    if (i == j) s = charge_exp[i] * SQRT2_PI / switch_fun[i];
    matrix_s[i*n+j] = s;

    if (matrix_d != NULL){
        double nrij = 0.0;
        nrij += coords[3*i]   * norm_vec[3*j];
        nrij += coords[3*i+1] * norm_vec[3*j+1];
        nrij += coords[3*i+2] * norm_vec[3*j+2];

        nrij -= coords[3*j]   * norm_vec[3*j];
        nrij -= coords[3*j+1] * norm_vec[3*j+1];
        nrij -= coords[3*j+2] * norm_vec[3*j+2];

        double rij2 = rij*rij;
        double rij3 = rij2*rij;
        double xi_r2_ij = xi_r_ij * xi_r_ij;
        double d = s * nrij / rij2 - 2.0*xi_r_ij/SQRT_PI*exp(-xi_r2_ij)*nrij/rij3;
        if (i == j) d = -charge_exp[i] * SQRT2_PI / (2.0*r_vdw[i]);
        matrix_d[i*n+j] = d;
    }
}

__global__
static void _pcm_dD_dS(double *matrix_dd, double *matrix_ds,
                    const double *coords, const double *norm_vec, const double *r_vdw,
                    const double *charge_exp, const double *switch_fun,
                    int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n || j >= n){
        return;
    }

    // calculate xi
    double ei = charge_exp[i];
    double ej = charge_exp[j];
    double xi_ij = ei * ej / sqrt(ei*ei + ej*ej);

    // calculate r
    double dx = coords[3*i]   - coords[3*j];
    double dy = coords[3*i+1] - coords[3*j+1];
    double dz = coords[3*i+2] - coords[3*j+2];
    double rij = norm3d(dx, dy, dz);

    double xi_r_ij = xi_ij * rij;
    double xi_r2_ij = xi_r_ij * xi_r_ij;
    if (i == j) rij = 1.0;
    double rij2 = rij*rij;

    double dS_dr = -(erf(xi_r_ij) -  2.0*xi_r_ij/ SQRT_PI * exp(-xi_r2_ij)) / rij2;
    if (i == j) dS_dr = 0.0;
    double dx_rij = dx / rij;
    double dy_rij = dy / rij;
    double dz_rij = dz / rij;

    matrix_ds[3*(i*n+j)]   = dS_dr * dx_rij;
    matrix_ds[3*(i*n+j)+1] = dS_dr * dy_rij;
    matrix_ds[3*(i*n+j)+2] = dS_dr * dz_rij;

    if (matrix_dd != NULL){
        double rij3 = rij2*rij;
        double nj_rij = dx*norm_vec[3*j] + dy*norm_vec[3*j+1] + dz*norm_vec[3*j+2];
        double dD_dri = 4.0*xi_r2_ij*xi_ij / SQRT_PI*exp(-xi_r2_ij)*nj_rij/rij3;
        if (i == j) dD_dri = 0.0;

        double nj_x = norm_vec[3*j];
        double nj_y = norm_vec[3*j+1];
        double nj_z = norm_vec[3*j+2];

        matrix_dd[3*(i*n+j)]   = dD_dri*dx_rij + dS_dr*(-nj_x/rij + 3.0*nj_rij/rij2*dx_rij);
        matrix_dd[3*(i*n+j)+1] = dD_dri*dy_rij + dS_dr*(-nj_y/rij + 3.0*nj_rij/rij2*dy_rij);
        matrix_dd[3*(i*n+j)+2] = dD_dri*dz_rij + dS_dr*(-nj_z/rij + 3.0*nj_rij/rij2*dz_rij);
    }
}

extern "C" {
int pcm_d_s(hipStream_t stream, double *matrix_d, double *matrix_s,
                    const double *coords, const double *norm_vec, const double *r_vdw,
                    const double *charge_exp, const double *switch_fun,
                    int n)
{
    int ntilex = (n + THREADS - 1) / THREADS;
    int ntiley = (n + THREADS - 1) / THREADS;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ntilex, ntiley);
    _pcm_d_s<<<blocks, threads, 0, stream>>>(matrix_d, matrix_s, coords, norm_vec, r_vdw, charge_exp, switch_fun, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}

int pcm_dd_ds(hipStream_t stream, double *matrix_dD, double *matrix_dS,
                    const double *coords, const double *norm_vec, const double *r_vdw,
                    const double *charge_exp, const double *switch_fun,
                    int n)
{
    int ntilex = (n + THREADS - 1) / THREADS;
    int ntiley = (n + THREADS - 1) / THREADS;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ntilex, ntiley);
    _pcm_dD_dS<<<blocks, threads, 0, stream>>>(matrix_dD, matrix_dS, coords, norm_vec, r_vdw, charge_exp, switch_fun, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}
}
